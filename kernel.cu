#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <math_functions.h>

#include "kernel.h"
__device__ float squarederror(float x[])
{
	float res=0;
	for(int i=0;i<NUM_OF_DIMENSIONS - 1 ; i++)
	{
		res+=x[i]*x[i];
	}
	return res;
}

__device__ float fitness_function(float x[])
{
    float res = 0;
    float y1 = 1 + (x[0] - 1) / 4;
    float yn = 1 + (x[NUM_OF_DIMENSIONS - 1] - 1) / 4;

    res += pow(sin(phi * y1), 2);

    for (int i = 0; i < NUM_OF_DIMENSIONS - 1; i++)
    {
        float y = 1 + (x[i] - 1) / 4;
        float yp = 1 + (x[i + 1] - 1) / 4;

        res += pow(y - 1, 2) * (1 + 10 * pow(sin(phi * yp), 2)) 
                + pow(yn - 1, 2);
    }

    return res;
}
__global__ void min(float * input, float *output, int len) {
  __shared__ float shared[32];
  int idx = blockIdx.x*blockDim.x+threadIdx.x;
  int cur = threadIdx.x;
  __syncthreads();
  if(idx<len)
  {
  	shared[cur] = input[idx];
  }
  else
  	shared[cur]=0;
  __syncthreads();
  //@@ Load a segment of the input vector into shared memory
  //@@ Traverse the reduction tree
  int s = 16;
  while(s>0)
  {
    if(cur<s && cur%NUM_OF_DIMENSIONS==0)
    {
      	if(fitness_function(shared+cur)<fitness_function(shared+cur+s))
		{
			for(int i=0;i<NUM_OF_DIMENSIONS;i++)
			{
				shared[cur+i]=shared[cur+s+i];
			}
		}
	}

	
    s = s/2;
    __syncthreads();
  }
  if(cur == 0)
{
	for(int i=0;i<NUM_OF_DIMENSIONS;i++)
	{
    	output[blockIdx.x+i] = shared[i];
	}
}
  __syncthreads();
  //@@ Write the computed sum of the block to the output vector at the
  //@@ correct index

}

__device__ float tempParticle1[NUM_OF_DIMENSIONS];
__device__ float tempParticle2[NUM_OF_DIMENSIONS];

// Fungsi yang dioptimasi
// Levy 3-dimensional

__global__ void kernelUpdateParticle(float *positions, float *velocities, 
                                     float *pBests, float *gBest, float r1, 
                                     float r2)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    
    if(i >= NUM_OF_PARTICLES * NUM_OF_DIMENSIONS)
        return;

    //float rp = getRandomClamped();
    //float rg = getRandomClamped();
    
    float rp = r1;
    float rg = r2;

    velocities[i] = OMEGA * velocities[i] + c1 * rp * (pBests[i] - positions[i])
            + c2 * rg * (gBest[i % NUM_OF_DIMENSIONS] - positions[i]);

    // Update posisi particle
    positions[i] += velocities[i];
}

__global__ void kernelUpdatePBest(float *positions, float *pBests, float* gBest)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    
    if(i >= NUM_OF_PARTICLES * NUM_OF_DIMENSIONS || i % NUM_OF_DIMENSIONS != 0)
        return;

    for (int j = 0; j < NUM_OF_DIMENSIONS; j++)
    {
        tempParticle1[j] = positions[i + j];
        tempParticle2[j] = pBests[i + j];
    }

    if (fitness_function(tempParticle1) < fitness_function(tempParticle2))
    {
        for (int k = 0; k < NUM_OF_DIMENSIONS; k++)
            pBests[i + k] = positions[i + k];
    }
}


extern "C" void cuda_pso(float *positions, float *velocities, float *pBests, 
                         float *gBest)
{
    int size = NUM_OF_PARTICLES * NUM_OF_DIMENSIONS;
    
    float *devPos;
    float *devVel;
    float *devPBest;
    float *devGBest;
    float *devOutput;
    
    float temp[NUM_OF_DIMENSIONS];
        
    // Memory allocation
    hipMalloc((void**)&devPos, sizeof(float) * size);
    hipMalloc((void**)&devVel, sizeof(float) * size);
    hipMalloc((void**)&devPBest, sizeof(float) * size);
    hipMalloc((void**)&devGBest, sizeof(float) * NUM_OF_DIMENSIONS);
    
    // Thread & Block number
    int threadsNum = 32;
    int blocksNum = NUM_OF_PARTICLES / threadsNum;
  	float *hostOutput = (float *)malloc(NUM_OF_DIMENSIONS*blocksNum  * sizeof(float));
    hipMalloc((void**)&devOutput, sizeof(float) * NUM_OF_DIMENSIONS*blocksNum);
    
    // Copy particle datas from host to device
    hipMemcpy(devPos, positions, sizeof(float) * size, hipMemcpyHostToDevice);
    hipMemcpy(devVel, velocities, sizeof(float) * size, 
               hipMemcpyHostToDevice);
    hipMemcpy(devPBest, pBests, sizeof(float) * size, hipMemcpyHostToDevice);
    hipMemcpy(devGBest, gBest, sizeof(float) * NUM_OF_DIMENSIONS, 
               hipMemcpyHostToDevice);
    
    // PSO main function
    for (int iter = 0; iter < MAX_ITER; iter++)
    {     
        // Update position and velocity
        kernelUpdateParticle<<<blocksNum, threadsNum>>>(devPos, devVel, 
                                                        devPBest, devGBest, 
                                                        getRandomClamped(), 
                                                        getRandomClamped());  
        // Update pBest
        kernelUpdatePBest<<<blocksNum, threadsNum>>>(devPos, devPBest, 
                                                     devGBest);
		min<<<blocksNum, threadsNum>>>(devPBest, devOutput, size);
        
        // Update gBest
        hipMemcpy(pBests, devPBest, 
                   sizeof(float) * NUM_OF_PARTICLES * NUM_OF_DIMENSIONS, 
                   hipMemcpyDeviceToHost);
        
        hipMemcpy(hostOutput, devOutput,sizeof(float)*NUM_OF_DIMENSIONS*blocksNum,hipMemcpyDeviceToHost);
        
        for(int i = 0; i < blocksNum*NUM_OF_DIMENSIONS; i += NUM_OF_DIMENSIONS)
        {
            for(int k = 0; k < NUM_OF_DIMENSIONS; k++)
                temp[k] = pBests[i + k];
        
            if (host_squarederror(temp) < host_squarederror(gBest))
            {
                for (int k = 0; k < NUM_OF_DIMENSIONS; k++)
                    gBest[k] = temp[k];
            }   
        }
        
        hipMemcpy(devGBest, gBest, sizeof(float) * NUM_OF_DIMENSIONS, 
                   hipMemcpyHostToDevice);
    }
    
    // Retrieve particle datas from device to host
    hipMemcpy(positions, devPos, sizeof(float) * size, hipMemcpyDeviceToHost);
    hipMemcpy(velocities, devVel, sizeof(float) * size, 
               hipMemcpyDeviceToHost);
    hipMemcpy(pBests, devPBest, sizeof(float) * size, hipMemcpyDeviceToHost);
    hipMemcpy(gBest, devGBest, sizeof(float) * NUM_OF_DIMENSIONS, 
               hipMemcpyDeviceToHost); 
    
    
    // cleanup
    hipFree(devPos);
    hipFree(devVel);
    hipFree(devPBest);
    hipFree(devGBest);
}

